#include "hip/hip_runtime.h"
/*
**    Programa para ilustrar el uso de los hilos y la memoria compartida.
**
**    Compilar con : nvcc --gpu-architecture=sm_50 cuenta_primos_gpu.cu -o primosGPU
**
**    Autor: Giovanny Alexis Reyes Vilchis
**    Matricula: 2163031812
*/

#include "../CUDA-by-Example/common/book.h"

/* Prototipo de las funciones */
__global__ void contarPrimos( int *enteros, int *siNo );
__device__ bool esPrimo(long n);
__host__ void llenarArreglo(int *arr, int tam);
__host__ void printArreglo(int *arr, int tam);

#define N (64 * 1024)

int main( void ) {
    /* Arreglos que tendrán los enteros que queremos verificar si son primos o no */
    int *enteros, *siNo;
    int *dev_enteros, *dev_siNo;

    /* Inicializa la semilla para generar números aleatorios */
    srand(time(NULL));

    printf("\nTamaño del arreglo: %d\n", N);

    // Asigna la memoria en la CPU
    enteros = (int *) malloc(N * sizeof(int));
    siNo = (int *) malloc(N * sizeof(int));

    /* El CPU debe mostrar el conteo final */
    printf("\nCPU: esperando resultados...\n");

    // Asigna la memoria en la GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_enteros, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_siNo, N * sizeof(int) ) );

    // Llena el arreglo 'enteros' en la CPU
    llenarArreglo(enteros, N);

    // Copia el arreglo 'enteros' a la GPU
    HANDLE_ERROR( hipMemcpy( dev_enteros, enteros, N * sizeof(int), hipMemcpyHostToDevice ) );

    contarPrimos<<<32,1024>>>( dev_enteros, dev_siNo );

    // Copia el arreglo 'siNo' de la GPU a la CPU
    HANDLE_ERROR( hipMemcpy( siNo, dev_siNo, N * sizeof(int), hipMemcpyDeviceToHost ) );

    //printArreglo(enteros, N);
    printf("\nLa GPU ya termino de identificar números primos.\n");
    //printArreglo(siNo, N);
    
    int numPrimos = 0;
    for (int i=0; i < N; i++)
        numPrimos += siNo[i];

    printf("\nHay %d primos.\n\n", numPrimos);

    // Libera la memoria que asignamos en la GPU
    HANDLE_ERROR( hipFree( dev_enteros ) );
    HANDLE_ERROR( hipFree( dev_siNo ) );

    // Libera la memoria que asignamos en la CPU
    free( enteros );
    free( siNo );

    return 0;
}

__global__ void contarPrimos( int *enteros, int *siNo ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < N)
    {
        if (esPrimo( enteros[tid] ) == true)
            siNo[tid] = true;
        else
            siNo[tid] = false;
    
        tid += blockDim.x * gridDim.x;
    }
}

__device__ bool esPrimo(long n) {
    bool primo = true;
    
    /* Buscar un divisor p entre 2 y n-1 para nuestro numero n*/
    long p = 2;
    while (primo == true && p < n-1) {
        if (n % p == 0)
            primo = false;

        p++;
    }
    
    return primo;
}

__host__ void llenarArreglo(int* arr, int tam) {
    for (int i=0; i < tam; i++) {
        arr[i] = 1000000 + rand()%10000;
        arr[i] = 2*arr[i] + 1;  // Para que el entero sea IMPAR.
    }
}

__host__ void printArreglo(int* arr, int tam) {
    for (int i=0; i < tam; i++)
        printf("%d ", arr[i]);
}
